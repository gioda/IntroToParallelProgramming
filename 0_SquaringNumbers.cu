/**
 * @Author: Giovanni Dalmasso <dalmasso>
 * @Date:   14-Sep-2018
 * @Email:  giovanni.dalmasso@embl.es
 * @Project: IntroToParallelProgramming
 * @Last modified by:   gioda
 * @Last modified time: 14-Sep-2018
 * @License: MIT
**/


#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void square(float *d_out, float *d_in)
{
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f*f;
}


int main(int argc, char ** argv)
{
    const int ARRAY_SIZE = 64;
    const int ARRAY_BITES = ARRAY_SIZE * sizeof(float);

    // generate the input array on the host
    float h_in[ARRAY_SIZE];
    for(int j=0; j<ARRAY_SIZE; j++)
    {
        h_in[j] = float(j);
    }
    float h_out[ARRAY_SIZE];

    // declare GPU memory pointers
    float * d_in;
    float * d_out;

    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BITES);
    hipMalloc((void **) &d_out, ARRAY_BITES);

    // transfer the array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BITES, hipMemcpyHostToDevice);

    // lauch the kernel
    square<<<1, ARRAY_SIZE>>>(d_out, d_in);

    // copy back the result array to the CPU
    hipMemcpy(h_out, d_out, ARRAY_BITES, hipMemcpyDeviceToHost);

    // print out the resulting array
    for(int j=0; j<ARRAY_SIZE; j++)
    {
        printf("%f", h_out[j]);
        printf(((j % 4) !=3) ? "\t" : "\n");
    }

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
